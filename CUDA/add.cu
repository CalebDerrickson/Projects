
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
    // Called a Grid-stride loop
    //https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }  
}

int main() 
{

    int N = 1<<20;

    float *x;
    float *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0;
        y[i] = 2.0;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check error
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout<<"Max Error: "<<maxError<<std::endl;

    hipFree(x);
    hipFree(y);

    return 0;
}